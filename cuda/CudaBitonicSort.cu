#include "hip/hip_runtime.h"
/* [CudaBitonicSort.cu]
 * author: Curt Bridgers
 * email: prestonbridgers@gmail.com
*/

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "helpers.h"
#include "wrappers.h"

#define ARR_MAX_INT 8192
#define DESCENDING 0
#define ASCENDING  1
#define MAX_THREADS 1024

__global__ void
d_bitonic_merge_kernel(int *arr, long arr_size, long local_size)
{
    long i;
    long tid = gridDim.x*blockIdx.x+threadIdx.x;
    long start_idx = tid * local_size;

    int half = local_size / 2;
    long end_idx = start_idx + half;
    int order = !(tid % 2);

    if (start_idx >= arr_size) return;

#ifdef DEBUG
    printf("[%ld] local_size: %ld\tstart: %ld\tend: %ld\torder: %d\n",
            tid, local_size, start_idx, end_idx + half - 1, order);
#endif
    for (i = start_idx; i < end_idx; i++) {
#ifdef DEBUG
        printf("[%ld] comparing: %d and %d\n",
               tid, arr[i], arr[i+half]);
#endif

        // Perform the swap if needed
        if (order == (arr[i] > arr[i+half])) {
            int tmp = arr[i];
            arr[i] = arr[i+half];
            arr[i+half] = tmp;
        }
#ifdef DEBUG
        printf("[%ld] After Swap: %d and %d\n",
               tid, arr[i], arr[i+half]);
#endif
    }

    // Split and sort some more :)
}


/* Jumping off function to run the bitonic sort kernels.

   arr  - The array to be sorted.
   size - The size (number of elements) in the array.
 */
void
bitonic_sort(int *arr, long size)
{
    long num_elems_per_subarray = 1;
    long num_subarrays = size / num_elems_per_subarray;
    long num_threads = num_subarrays / 2;
    long stage = 1;

    // Copying array to cuda device
    int *d_arr;
    hipMalloc((void**)&d_arr, size * sizeof(int));
    hipMemcpy(d_arr, arr, size * sizeof(int), H2D);
    CUERR;

    while (num_elems_per_subarray != size)
    {
        /* printf("num_elems_per_subarray: %d\nnum_blocks: %d\nnum_threads: %d\n\n", */
        /*         num_elems_per_subarray, num_subarrays, num_threads); */
#ifdef DEBUG
        printf("\n~~~~~~~~~~~~~~~~~~~~~~~Stage %ld~~~~~~~~~~~~~~~~~~~~~~~~~~~~~\n",
                stage);
#endif

        // Call kernel with grid=1,1,1 block=num_threads,1,1
        // Each thread in the block will have 2 subarrays to merge
        int num_blocks = SDIV(num_threads, MAX_THREADS);
        dim3 grid(num_blocks,1,1);
        dim3 block(MAX_THREADS,1,1);
#ifdef DEBUG
        printf("grid(%d)\tblock(%d)\n", num_blocks, MAX_THREADS);
#endif
        d_bitonic_merge_kernel<<<grid, block>>>(d_arr, size, 2*num_elems_per_subarray);
        CUERR;

#ifdef DEBUG
        usleep(500000);
#endif

        num_elems_per_subarray *= 2;
        num_subarrays = size / num_elems_per_subarray;
        num_threads = num_subarrays / 2;
        stage += 1;
    }

    hipMemcpy(arr, d_arr, size * sizeof(int), D2H);
    CUERR;
    hipFree(d_arr);
}

/* Initializes a given integer array of a given size to random numbers
 * in the range [0,ARR_MAX_INT).
 *
 * arr  - A pointer to an integer array that has been allocated size*4 bytes
 *        of memory.
 * size - The size of the array (number of elements).
 */
void
init_array(int *arr, const long size) {
    int i;

    for (i = 0; i < size; i++) {
        /* arr[i] = rand() % ARR_MAX_INT; */
        arr[i] = i;
    }
}

/* Pretty prints the contents of an integer array.
 *
 * arr   - The array to print.
 * size  - The size of the array (number of elements).
 * label - A header string that will be printed above the contents of the
 *         array.
 */
void
print_array(int *arr, const long size, char *label) {
    int i;

    printf("%s:\n[", label);
    for (i = 0; i < size; i++) {
        if (i == size - 1) {
            printf("%d]\n", arr[i]);
        } else {
            printf("%d, ", arr[i]);
        }
    }
    return;
}

/* Prints the usage of the program at the command line and exits the program.
 */
void
print_usage(int argc, char **argv)
{
    printf("usage: %s array_size\n", argv[0]);
    printf("  array_size: The given number, n, will result in an array of size"
           " 2^n elements (n must be larger than 0).\n");
    exit(1);
}

/* Cuda Bitonic Sort
 */
int
main(int argc, char *argv[])
{
    int *arr;
    int arr_size;
    int exponent;

    // Check arguments
    if (argc < 2) print_usage(argc, argv);
    exponent = atoi(argv[1]);
    if (exponent == 0) print_usage(argc, argv);

    // Seed random number generator
	srand(time(NULL));

    // Allocate and initialize the array
    arr_size = pow(2, exponent);
    arr = (int*) malloc(arr_size * sizeof(*arr));
    init_array(arr, arr_size);

#ifdef DEBUG
    char label1[] = "Before";
    print_array(arr, arr_size, label1);
#endif

    // Perform the sort
    bitonic_sort(arr, arr_size);

#ifdef DEBUG
    char label2[] = "\nAfter";
    print_array(arr, arr_size, label2);
#endif

	return EXIT_SUCCESS;	
}

